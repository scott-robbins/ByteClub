
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add (int*a, int*b, int*c) {
        *c = *a + *b;
}

int main(void){
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    // Make space for device copies of vars
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = 2;
    b = 7;

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);// Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    // show the result
    printf("Result: %d\n", c);
    return 0;
}