#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add (int*a, int*b, int*c) {
        *c = *a + *b;
}

void random_ints(int* arr, int n){
    int i;
    for (i = 0; i < n; ++i)
        arr[i] = rand();
}

#define N 512
int main(void){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    // Allocate space for device copies of a,b and c
    hipMalloc((void **), &d_a, size);
    hipMalloc((void **), &d_b, size);
    hipMalloc((void **), &d_c, size);

    // Do the same for host copy space
    a = (int *)malloc(size);  random_ints(a, N);
    b = (int *)malloc(size);  random_ints(b, N);
    c = (int *)malloc(size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with N blocks
    add<<<N,1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyHostToDevice);

    // Cleanup memory
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}